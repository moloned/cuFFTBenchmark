#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <iostream>

#define C2R 1
#define R2C 2
#define C2C 3
#define Z2D 5
#define D2Z 6
#define Z2Z 7
#define _FROMTO FROMTO

#if _FROMTO == Z2Z
#define TO_TYPE hipfftDoubleComplex
#define FROM_TYPE hipfftDoubleComplex
#define FROMTO_STR "double precision complex-to-complex"
#elif _FROMTO == D2Z
#define TO_TYPE hipfftDoubleComplex
#define FROM_TYPE hipfftDoubleReal
#define FROMTO_STR "double precision real-to-complex"
#elif _FROMTO == Z2D
#define TO_TYPE hipfftDoubleReal
#define FROM_TYPE hipfftDoubleComplex
#define FROMTO_STR "double precision complex-to-real"
#elif _FROMTO == C2C
#define TO_TYPE hipfftComplex
#define FROM_TYPE hipfftComplex
#define FROMTO_STR "single precision complex-to-complex"
#elif _FROMTO == R2C
#define TO_TYPE hipfftComplex
#define FROM_TYPE hipfftReal
#define FROMTO_STR "single precision real-to-complex"
#elif _FROMTO == C2R
#define TO_TYPE hipfftReal
#define FROM_TYPE hipfftComplex
#define FROMTO_STR "single precision complex-to-real"
#else
#error "FROMTO must be one of Z2Z, Z2D, D2Z, C2C, R2C and C2R"
#endif
template <class A, class B>
hipfftResult_t CUFFTPLAN2D(hipfftHandle *plan, int size_x, int size_y, A* in, B* out);

hipfftResult_t CUFFTPLAN2D( hipfftHandle *plan, int size_x, int size_y, 
                     hipfftDoubleComplex* in, hipfftDoubleComplex* out) {
      return hipfftPlan2d(plan, size_x, size_y, HIPFFT_Z2Z);
}

hipfftResult_t CUFFTPLAN2D( hipfftHandle *plan, int size_x, int size_y, 
                     hipfftDoubleReal* in, hipfftDoubleComplex* out) {
      return hipfftPlan2d(plan, size_x, size_y, HIPFFT_D2Z);
}

hipfftResult_t CUFFTPLAN2D( hipfftHandle *plan, int size_x, int size_y, 
                     hipfftDoubleComplex* in, hipfftDoubleReal* out) {
      return hipfftPlan2d(plan, size_x, size_y, HIPFFT_Z2D);
}

hipfftResult_t CUFFTPLAN2D( hipfftHandle *plan, int size_x, int size_y, 
                     hipfftComplex* in, hipfftComplex* out) {
      return hipfftPlan2d(plan, size_x, size_y, HIPFFT_C2C);
}

hipfftResult_t CUFFTPLAN2D( hipfftHandle *plan, int size_x, int size_y, 
                     hipfftReal* in, hipfftComplex* out) {
      return hipfftPlan2d(plan, size_x, size_y, HIPFFT_R2C);
}

hipfftResult_t CUFFTPLAN2D( hipfftHandle *plan, int size_x, int size_y, 
                     hipfftComplex* in, hipfftReal* out) {
      return hipfftPlan2d(plan, size_x, size_y, HIPFFT_C2R);
}

template <class A, class B>
hipfftResult_t CUFFTEXEC(hipfftHandle plan, A* in, B* out); 

hipfftResult_t CUFFTEXEC (
                             hipfftHandle plan, hipfftDoubleComplex* in, hipfftDoubleComplex* out) {
      return hipfftExecZ2Z(plan, in, out, HIPFFT_FORWARD);
}

hipfftResult_t CUFFTEXEC(
                             hipfftHandle plan, hipfftDoubleReal* in, hipfftDoubleComplex* out) {
      return hipfftExecD2Z(plan, in, out);
}

hipfftResult_t CUFFTEXEC(
                             hipfftHandle plan, hipfftDoubleComplex* in, hipfftDoubleReal* out) {
      return hipfftExecZ2D(plan, in, out);
}

hipfftResult_t CUFFTEXEC(
                             hipfftHandle plan, hipfftComplex* in, hipfftComplex* out) {
      return hipfftExecC2C(plan, in, out, HIPFFT_FORWARD);
}

hipfftResult_t CUFFTEXEC(
                             hipfftHandle plan, hipfftReal* in, hipfftComplex* out) {
      return hipfftExecR2C(plan, in, out);
}

hipfftResult_t CUFFTEXEC(
                             hipfftHandle plan, hipfftComplex* in, hipfftReal* out) {
      return hipfftExecC2R(plan, in, out);
}

int main(void) {

  int NX=10112, NY=10112;
  int size = NX*NY;
  float elapsed;
  hipfftHandle plan;
  FROM_TYPE *data1;
  hipMalloc(&data1, sizeof(FROM_TYPE)*NX*NY);
#ifndef INPLACE
  TO_TYPE *data2;
  hipMalloc(&data2, sizeof(TO_TYPE)*NX*NY);
#endif

  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);

  hipError_t err;
  err = hipGetLastError();
  if (err) std::cout << "Error in initial copy" << std::endl;
  std::cin >> NX >> NY;
  std::cout << "**** " << FROMTO_STR << " ****" << std::endl;
  std::cout << "dx, dy, elapsed, Gcell/s, Gflps" << std::endl; 
#ifdef INPLACE 
#define TARGET data1
#else
#define TARGET data2
#endif
  while( NX != 0) {
    hipfftResult_t r = CUFFTPLAN2D(&plan, NX, NY, data1, TARGET);
    hipEventRecord(start);
    for (int z=0; z< 5; z++)
       if (!r) r = CUFFTEXEC(plan, data1, TARGET);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    hipError_t err;
    err = hipGetLastError();
    if (err) std::cout << NX << ", " << NY << " - Error " << err <<" : " <<
                        hipGetErrorString(err) << std::endl;
    else if (r) std::cout << NX << ", " << NY << " - CUFFT Error " << r << 
                        std::endl;
    else std::cout << NX << ", " << NY << ", " << elapsed/5 << ", " 
              << 5*NX*NY/elapsed/1000/1000 << ", " << 5*5/elapsed/1000/1000*NX*NY*(log2(NX+0.000)+log2(NY+0.000)) << std::endl;
    hipfftDestroy(plan);
    std::cin >> NX >> NY;
    if (NX*NY > size) {
       std::cout << "Reallocating to " << NX << " x " << NY << std::endl;
       hipFree(data1); data1=0;
       hipMalloc(&data1, sizeof(hipfftDoubleComplex)*NX*NY);
       if(!data1) std::cout << "Failed to allocate data1!" << std::endl;
#ifndef INPLACE
       hipFree(data2); data2=0;
       hipMalloc(&data2, sizeof(hipfftDoubleComplex)*NX*NY);
       if(!data2) std::cout << "Failed to allocate data2!\n" << std::endl;
#endif
       size = NX*NY;
    }
  }
  std::cout << "0, 0" << std::endl;
 
  //printf("(%d,%d) - Error %d: %s\n", NX, NY, err, hipGetErrorString(err));

  hipFree(data1); 
#ifndef INPLACE
  hipFree(data2);
#endif
  hipEventDestroy(start); hipEventDestroy(stop);


  return 0;
}
